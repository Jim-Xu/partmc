/* Copyright (C) 2015-2018 Matthew Dawson
 * Licensed under the GNU General Public License version 2 or (at your
 * option) any later version. See the file COPYING for details.
 *
 * GPU solver functions
 *
*/
/** \file
 * \brief GPU solver functions
*/
#include "cuda_util.h"
extern "C" {
#include "aero_rep_solver.h"
#include "phlex_gpu_solver.h"
#include "rxn_gpu_solver.h"
#include "rxn_solver.h"
#include "sub_model_solver.h"
}

// State variable types (Must match parameters defined in pmc_chem_spec_data module)
#define CHEM_SPEC_UNKNOWN_TYPE 0
#define CHEM_SPEC_VARIABLE 1
#define CHEM_SPEC_CONSTANT 2
#define CHEM_SPEC_PSSA 3
#define CHEM_SPEC_ACTIVITY_COEFF 4

/** Create a new set of GPU solver data and add it to a SolverData object
 *
 * \param solver_data Solver data
 */
extern "C"
void phlex_gpu_solver_new( SolverData *solver_data )
{

  // Determine the current GPU memory usage
  size_t free_mem, total_mem;
  hipMemGetInfo( &free_mem, &total_mem );
  printf("\nINFO (PartMC-8351420968): GPU total memory %zu, GPU free memory %zu\n",
    total_mem, free_mem);

  // Create a new SolverDeviceData object
  solver_data->solver_device_data = ( void * )
                                   malloc( sizeof( SolverDeviceData ) );
  if( solver_data->solver_device_data == NULL ) {
    printf("\n\nERROR allocating space for SolverDeviceData\n\n");
    exit( 1 );
  }

  // Get a pointer to the SolverDeviceData object
  SolverDeviceData *sdd = ( SolverDeviceData * ) 
                          solver_data->solver_device_data;
  
  // Save the number of states to solve
  sdd->n_states = solver_data->n_states;

  // Allocate the ModelDeviceData array
  HANDLE_ERROR( hipHostAlloc( ( void** ) &( sdd->host_model_dev_data ),
                               sdd->n_states * sizeof( ModelDeviceData ),
                               hipHostMallocWriteCombined |
                                  hipHostMallocMapped
                             ) );
  HANDLE_ERROR( hipHostGetDevicePointer(
                               (void**) &( sdd->dev_model_dev_data ),
                               (void*) sdd->host_model_dev_data,
                               0
                             ) );

  // Start solving the first state on the first block
  sdd->deriv_threads     = 0;
  sdd->jac_threads       = 0;
  sdd->env_threads       = 0;
  int curr_deriv_block   = 0;
  int curr_jac_block     = 0;
  int curr_env_block     = 0;
  int curr_deriv_threads = 0;
  int curr_jac_threads   = 0;
  int curr_env_threads   = 0;
  int curr_deriv_id      = 0;
  int curr_jac_id        = 0;

  // Set up a ModelDeviceData object for each ModelData object
  for( int i_state = 0; i_state < sdd->n_states; i_state++ ) {

    // Get a pointer to the ModelData object
    ModelData * md = &( solver_data->model_data[ i_state ] );

    // Get a pointer to the device data object
    ModelDeviceData * mdd = &( sdd->host_model_dev_data[ i_state ] );

    // Set up the working state array
    HANDLE_ERROR( hipHostAlloc( ( void** ) &( mdd->host_state ), 
                                 md->n_state_var * sizeof( PMC_C_FLOAT ),
                                 hipHostMallocWriteCombined |
                                    hipHostMallocMapped
                               ) );
    HANDLE_ERROR( hipHostGetDevicePointer( 
                                 (void**) &(mdd->dev_state), 
                                 (void*) mdd->host_state,
                                 0
                               ) );

    // Set up the working environmental array
    HANDLE_ERROR( hipHostAlloc( (void**) &(mdd->host_env), 
                                 md->n_env_var * sizeof(PMC_C_FLOAT),
                                 hipHostMallocWriteCombined |
                                    hipHostMallocMapped
                               ) );
    HANDLE_ERROR( hipHostGetDevicePointer( 
                                 (void**) &(mdd->dev_env), 
                                 (void*) mdd->host_env,
                                 0
                               ) );

    // Set up the working derivative array
    HANDLE_ERROR( hipHostAlloc( (void**) &(mdd->host_deriv),
                                 md->deriv_size * sizeof(PMC_SOLVER_C_FLOAT),
                                 hipHostMallocMapped
                               ) );
    HANDLE_ERROR( hipHostGetDevicePointer( 
                                 (void**) &(mdd->dev_deriv),
                                 (void*) mdd->host_deriv,
                                 0
                               ) );
    mdd->deriv_size = md->deriv_size;

    // Set up the working Jacobian data array
    HANDLE_ERROR( hipHostAlloc( (void**) &(mdd->host_jac),
                                 ( md->jac_size > 0 ? md->jac_size : 1 ) 
                                      * sizeof(PMC_SOLVER_C_FLOAT),
                                 hipHostMallocMapped
                               ) );
    HANDLE_ERROR( hipHostGetDevicePointer( 
                                 (void**) &(mdd->dev_jac),
                                 (void*) mdd->host_jac,
                                 0
                               ) );
    mdd->jac_size = md->jac_size;

    // Initialize the reaction data
    rxn_gpu_solver_new( mdd, md->rxn_data );

    ////////////////////////////
    // Set ids for this state //
    ////////////////////////////
    
    int n_rxn = ( (RxnDeviceData*)(mdd->host_rxn_dev_data) )->n_rxn;
    
    // Advance to the next block once the deriv or jac array is too large 
    if( curr_deriv_id + mdd->deriv_size > MAX_SHARED_ARRAY_SIZE_ ||
        curr_deriv_threads + n_rxn > CUDA_MAX_THREADS ) {
      curr_deriv_block++;
      if( curr_deriv_threads > sdd->deriv_threads ) 
        sdd->deriv_threads = curr_deriv_threads;
      curr_deriv_threads = curr_deriv_id = 0;
    }
    if( curr_jac_id + mdd->jac_size > MAX_SHARED_ARRAY_SIZE_ ||
        curr_jac_threads + n_rxn > CUDA_MAX_THREADS ) {
      curr_jac_block++;
      if( curr_jac_threads > sdd->jac_threads ) 
        sdd->jac_threads = curr_jac_threads;
      curr_jac_threads = curr_jac_id = 0;
    }
    if( curr_env_threads + n_rxn > CUDA_MAX_THREADS ) {
      curr_env_block++;
      if( curr_env_threads > sdd->env_threads )
        sdd->env_threads = curr_env_threads;
      curr_env_threads = 0;
    }

    // Set the current state ids
    mdd->deriv_block    = curr_deriv_block;
    mdd->jac_block      = curr_jac_block;
    mdd->env_block      = curr_env_block;
    mdd->deriv_start_id = curr_deriv_id;
    mdd->jac_start_id   = curr_jac_id;

    // Advance the number of threads and array elements
    curr_deriv_threads += n_rxn;
    curr_jac_threads   += n_rxn;
    curr_env_threads   += n_rxn;
    curr_deriv_id      += mdd->deriv_size;
    curr_jac_id        += mdd->jac_size;
  }

  if( curr_deriv_threads > sdd->deriv_threads )
    sdd->deriv_threads = curr_deriv_threads;
  if( curr_jac_threads > sdd->jac_threads )
    sdd->jac_threads   = curr_jac_threads;
  if( curr_env_threads > sdd->env_threads )
    sdd->env_threads   = curr_env_threads;
  sdd->deriv_blocks    = curr_deriv_block + 1;
  sdd->jac_blocks      = curr_jac_block + 1;
  sdd->env_blocks      = curr_env_block + 1;
}

/** \brief Update the environmental state
 *
 * \param SolverData Solver data
 */
extern "C"
void phlex_gpu_solver_update_env_state( SolverData *solver_data )
{
  SolverDeviceData * sdd = ( SolverDeviceData * )
                           solver_data->solver_device_data;

  // Update the environmental state for GPU functions
  for( int i_state = 0; i_state < solver_data->n_states; i_state++ ) {
    ModelData * md = &( solver_data->model_data[ i_state ] );
    ModelDeviceData * mdd = &( sdd->host_model_dev_data[ i_state ] );
    for( int i_var = 0; i_var < md->n_env_var; i_var++ ) {
      mdd->host_env[ i_var ] = md->env[ i_var ];
    }
  }

  // Update the environmental state for reactions with GPU functions
  dim3 dimGrid( sdd->env_blocks );
  dim3 dimBlock( sdd->env_threads );
  rxn_gpu_update_env_state<<< dimGrid, dimBlock >>>( *sdd );
  hipDeviceSynchronize();

  // Update the remaining reactions
  for( int i_state = 0; i_state < solver_data->n_states; i_state++ ) {
    ModelData * md = &( solver_data->model_data[ i_state ] );
    rxn_update_env_state( *md );
  }

}

/** \brief Compute the time derivative f(t,y)
 *
 * \param t Current model time (s)
 * \param y Dependent variable array
 * \param deriv Time derivative vector f(t,y) to calculate
 * \param solver_data Pointer to the solver data
 * \return Status code
 */
extern "C"
int phlex_gpu_solver_f( realtype t, N_Vector y, N_Vector deriv,
          void *solver_data )
{
  SolverData *sd = (SolverData*) solver_data;
  SolverDeviceData *sdd = ( SolverDeviceData* ) ( sd->solver_device_data );
  realtype time_step;

  // Loop through the states to solve
  for( int i_dep_var = 0, i_state = 0; i_state < sd->n_states; i_state++ ) {
    
    // Get pointers to the ModelData and ModelDeviceData for this state
    ModelData *md = &( sd->model_data[ i_state ] );
    ModelDeviceData *mdd = &( sdd->host_model_dev_data[ i_state ] );

    // Update the state array with the current dependent variable values
    // Signal a recoverable error (positive return value) for negative 
    // concentrations.
    for( int j_spec = 0, j_dep_var = 0; j_spec < md->n_state_var; j_spec++ ) {
      if( md->var_type[ j_spec ] == CHEM_SPEC_VARIABLE ) {
        if( NV_DATA_S( y )[ i_dep_var ] < 0.0 ) return 1;
        md->state[ j_spec ] = 
                ( PMC_C_FLOAT ) ( NV_DATA_S( y )[ i_dep_var++ ] );
        mdd->host_deriv[ j_dep_var++ ] = 0.0;     
      }
    }

    // Update the aerosol representations
    aero_rep_update_state( *md );

    // Run the sub models
    sub_model_calculate( *md );

    // Run pre-derivative calculations
    rxn_pre_calc( *md );

    // Update the state array for use on the GPUs
    for( int j_spec = 0; j_spec < md->n_state_var; j_spec++ )
        mdd->host_state[ j_spec ] = md->state[ j_spec ];

  }

  // Get the current integrator time step (s)
  CVodeGetCurrentStep( sd->cvode_mem, &time_step );
  
  // Calculate the time derivative f(t,y) for GPU rxns
  dim3 dimGrid( sdd->deriv_blocks );
  dim3 dimBlock( sdd->deriv_threads );
  rxn_gpu_calc_deriv<<< dimGrid, dimBlock >>>( *sdd, (PMC_C_FLOAT) time_step );
  hipDeviceSynchronize();

  // Loop through the states to solve
  for( int i_dep_var = 0, i_state = 0; i_state < sd->n_states; i_state++ ) {
  
    // Get pointers to the ModelData and ModelDeviceData for this state
    ModelData *md = &( sd->model_data[ i_state ] );
    ModelDeviceData *mdd = &( sdd->host_model_dev_data[ i_state ] );

    // Calculate the remaining time derivatives f(t,y)
    rxn_calc_deriv( *md, mdd->host_deriv, (PMC_C_FLOAT) time_step );
  
    // Copy working derivative array to solver derivative
    for( int i_spec = 0; i_spec < md->deriv_size; i_spec++ )
          NV_DATA_S( deriv )[ i_dep_var++ ] = ( realtype ) mdd->host_deriv[ i_spec ];
  
  }

  return (0);

}

/** \brief Compute the Jacobian
 *
 * \param t Current model time (s)
 * \param y Dependent variable array
 * \param deriv Time derivative vector f(t,y)
 * \param J Jacobian to calculate
 * \param solver_data Pointer to the solver data
 * \param tmp1 Unused vector
 * \param tmp2 Unused vector
 * \param tmp3 Unused vector
 * \return Status code
 */
int phlex_gpu_solver_Jac( realtype t, N_Vector y, N_Vector deriv, SUNMatrix J,
        void *solver_data, N_Vector tmp1, N_Vector tmp2, N_Vector tmp3 )
{
  SolverData *sd = (SolverData*) solver_data;
  SolverDeviceData *sdd = ( SolverDeviceData* ) ( sd->solver_device_data );
  realtype time_step;

  // TODO Figure out how to keep the Jacobian from being redimensioned
  // Reset the Jacobian dimensions
  if ( SM_NNZ_S( J ) < SM_NNZ_S( sd->J_init ) ) {
    SM_INDEXVALS_S( J ) = ( sunindextype* ) realloc( SM_INDEXVALS_S( J ),
              SM_NNZ_S( sd->J_init ) * sizeof( sunindextype ) );
    if( SM_INDEXVALS_S( J ) == NULL ) {
      printf( "\n\nERROR allocating space for sparse matrix index values\n\n" );
      exit( 1 );
    }
    SM_DATA_S( J ) = ( realtype* ) realloc( SM_DATA_S( J ),
              SM_NNZ_S( sd->J_init ) * sizeof( realtype ) );
    if ( SM_DATA_S( J ) == NULL ) {
      printf( "\n\nERROR allocating space for sparse matrix data\n\n" );
      exit( 1 );
    }
  }
  SM_NNZ_S( J ) = SM_NNZ_S( sd->J_init );
  for( int i = 0; i < SM_NNZ_S( J ); i++ ) {
    ( SM_INDEXVALS_S( J ) )[ i ] = ( SM_INDEXVALS_S( sd->J_init ) )[ i ];
  }
  for( int i = 0; i <= SM_NP_S( J ); i++ ) {
    ( SM_INDEXPTRS_S( J ) )[ i ] = ( SM_INDEXPTRS_S( sd->J_init ) )[ i ];
  } 

  // Loop through the states to solve
  for( int i_dep_var = 0, i_state = 0; i_state < sd->n_states; i_state++ ) {
    
    // Get pointers to the ModelData and ModelDeviceData for this state
    ModelData *md = &( sd->model_data[ i_state ] );
    ModelDeviceData *mdd = &( sdd->host_model_dev_data[ i_state ] );

    // Reset the Jacobian data for this state
    for( int i_elem = 0; i_elem < md->jac_size; i_elem++ )
      mdd->host_jac[ i_elem ] = ( PMC_SOLVER_C_FLOAT ) ZERO;
  
    // Update the state array with the current dependent variable values
    // Signal a recoverable error (positive return value) for negative 
    // concentrations.
    for( int j_spec = 0; j_spec < md->n_state_var; j_spec++ ) {
      if( md->var_type[ j_spec ] == CHEM_SPEC_VARIABLE ) {
        if( NV_DATA_S( y )[ i_dep_var ] < 0.0 ) return 1;
        md->state[ j_spec ] = 
                ( PMC_C_FLOAT ) ( NV_DATA_S( y )[ i_dep_var++ ] );
      }
    }

    // Update the aerosol representations
    aero_rep_update_state( *md );

    // Run the sub models
    sub_model_calculate( *md );

    // Run pre-derivative calculations
    rxn_pre_calc( *md );

    // Update the state array for use on the GPUs
    for( int j_spec = 0; j_spec < md->n_state_var; j_spec++ )
        mdd->host_state[ j_spec ] = md->state[ j_spec ];

  }

  // Get the current integrator time step (s)
  CVodeGetCurrentStep(sd->cvode_mem, &time_step);
  
  // Calculate the Jacobian for GPU rxns
  dim3 dimGrid( sdd->jac_blocks );
  dim3 dimBlock( sdd->jac_threads );
  rxn_gpu_calc_jac<<< dimGrid, dimBlock >>>( *sdd, (PMC_C_FLOAT) time_step );
  hipDeviceSynchronize();

  // Loop through the states to solve
  for( int i_jac_elem = 0, i_state = 0; i_state < sd->n_states; i_state++ ) {
  
    // Get pointers to the ModelData and ModelDeviceData for this state
    ModelData *md = &( sd->model_data[ i_state ] );
    ModelDeviceData *mdd = &( sdd->host_model_dev_data[ i_state ] );

    // Calculate the Jacobian for the remaining rxns
    rxn_calc_jac( *md, mdd->host_jac, time_step );

    // Copy the working Jacobian back into the solver Jacobian
    for( int i = 0; i < md->jac_size; i++ )
      SM_DATA_S( J )[ i_jac_elem++ ] = ( realtype )( mdd->host_jac[ i ] );

  }

  return (0);

}

/** \brief Free GPU solver device data memory from a void pointer
  */
extern "C"
void phlex_gpu_solver_solver_device_data_free_vp( void * solver_device_data )
{
  SolverDeviceData *sd = ( SolverDeviceData * ) solver_device_data;
  phlex_gpu_solver_solver_device_data_free( *sd );
  free( sd );
}

/** \brief Free GPU solver device data memory
  */
extern "C"
void phlex_gpu_solver_solver_device_data_free( 
          SolverDeviceData solver_device_data )
{
  for( int i_state = 0; i_state < solver_device_data.n_states; i_state++ ) {
    phlex_gpu_solver_model_device_data_free( 
              solver_device_data.host_model_dev_data[ i_state ] );
  } 
  HANDLE_ERROR( hipHostFree( solver_device_data.host_model_dev_data ) );
}

/** \brief Free GPU model device data memory
  */
extern "C"
void phlex_gpu_solver_model_device_data_free(
          ModelDeviceData model_device_data )
{
  rxn_gpu_solver_free( model_device_data.host_rxn_dev_data );
  HANDLE_ERROR( hipHostFree( model_device_data.host_state ) );
  HANDLE_ERROR( hipHostFree( model_device_data.host_deriv ) );
  HANDLE_ERROR( hipHostFree( model_device_data.host_jac ) );
}
