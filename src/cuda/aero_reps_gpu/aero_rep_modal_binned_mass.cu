#include "hip/hip_runtime.h"
/* Copyright (C) 2019 Christian Guzman
 * Licensed under the GNU General Public License version 1 or (at your
 * option) any later version. See the file COPYING for details.
 *
 * Modal mass aerosol representation functions
 *
 */
/** \file
 * \brief Modal mass aerosol representation functions
 */
extern "C" {
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "../aeros_gpu/aero_phase_solver_gpu.h"
#include "../aeros_gpu/aero_reps_gpu.h"
#include "../camp_gpu_solver.h"

#define TEMPERATURE_K_ env_data[0]
#define PRESSURE_PA_ env_data[1]

#define UPDATE_GMD 0
#define UPDATE_GSD 1

#define BINNED 1
#define MODAL 2

#define NUM_SECTION_ (int_data[0])
#define INT_DATA_SIZE_ (int_data[1])
#define FLOAT_DATA_SIZE_ (int_data[2])
#define AERO_REP_ID_ (int_data[3])
#define NUM_INT_PROP_ 4
#define NUM_FLOAT_PROP_ 0
#define MODE_INT_PROP_LOC_(x) (int_data[NUM_INT_PROP_+x]-1)
#define MODE_FLOAT_PROP_LOC_(x) (int_data[NUM_INT_PROP_+NUM_SECTION_+x]-1)
#define SECTION_TYPE_(x) (int_data[MODE_INT_PROP_LOC_(x)])

// For modes, NUM_BINS_ = 1
#define NUM_BINS_(x) (int_data[MODE_INT_PROP_LOC_(x)+1])

// Number of aerosol phases in this mode/bin set
#define NUM_PHASE_(x) (int_data[MODE_INT_PROP_LOC_(x)+2])

// Phase state and model data ids
#define PHASE_STATE_ID_(x, y, b) (int_data[MODE_INT_PROP_LOC_(x)+3+b*NUM_PHASE_(x)+y]-1)
#define PHASE_MODEL_DATA_ID_(x, y, b) (int_data[MODE_INT_PROP_LOC_(x)+3+NUM_BINS_(x)*NUM_PHASE_(x)+b*NUM_PHASE_(x)+y]-1)

// GMD and bin diameter are stored in the same position - for modes, b=0
#define GMD_(x, b) (float_data[MODE_FLOAT_PROP_LOC_(x)+b*4])
#define BIN_DP_(x, b) (float_data[MODE_FLOAT_PROP_LOC_(x)+b*4])

// GSD - only used for modes, b=0
#define GSD_(x, b) (float_data[MODE_FLOAT_PROP_LOC_(x)+b*4+1])

// Real-time number concentration - used for modes and bins - for modes, b=0
#define NUMBER_CONC_(x, b) (float_data[MODE_FLOAT_PROP_LOC_(x)+b*4+2])

// Real-time effective radius - only used for modes, b=0
#define EFFECTIVE_RADIUS_(x, b) (float_data[MODE_FLOAT_PROP_LOC_(x)+b*4+3])

// Real-time phase mass (ug/m^3) - used for modes and bins - for modes, b=0
#define PHASE_MASS_(x, y, b) (float_data[MODE_FLOAT_PROP_LOC_(x)+4*NUM_BINS_(x)+b*NUM_PHASE_(x)+y])

// Real-time phase average MW (kg/mol) - used for modes and bins - for modes, b=0
#define PHASE_AVG_MW_(x, y, b) (float_data[MODE_FLOAT_PROP_LOC_(x)+(4+NUM_PHASE_(x))*NUM_BINS_(x)+b*NUM_PHASE_(x)+y])


/** \brief Get the effective particle radius \f$r_{eff}\f$ (m)
 *
 * The modal mass effective radius is calculated for a log-normal distribution
 * where the geometric mean diameter (\f$\tilde{D}_n\f$) and geometric standard
 * deviation (\f$\tilde{\sigma}_g\f$) are set by the aerosol model prior to
 * solving the chemistry. Thus, all \f$\frac{\partial r_{eff}}{\partial y}\f$
 * are zero. The effective radius is calculated according to the equation given
 * in Table 1 of Zender \cite Zender2002 :
 *
 * \f[
 *      r_{eff} = \frac{\tilde{D}_n}{2}*exp(5\tilde{\sigma}_g^2/2)
 * \f]
 * \f[
 *      r_{eff} = \frac{D_{eff}}{2}
 * \f]
 *
 * For bins, \f$r_{eff}\f$ is assumed to be the bin radius.
 *
 * \param aero_phase_idx Index of the aerosol phase within the representation
 * \param radius Effective particle radius (m)
 * \param partial_deriv \f$\frac{\partial r_{eff}}{\partial y}\f$ where \f$y\f$
 *                       are species on the state array
 * \param aero_rep_data Pointer to the aerosol representation data
 * \return The aero_rep_data pointer advanced by the size of the aerosol
 *         representation
 */
__device__ void *aero_rep_gpu_modal_binned_mass_get_effective_radius(int aero_phase_idx,
                                                      double *radius, double *partial_deriv, void *aero_rep_data) {
  int *int_data = (int *) aero_rep_data;
  double *float_data = (double *) &(int_data[INT_DATA_SIZE_]);

  for (int i_section = 0; i_section < NUM_SECTION_; i_section++) {
    for (int i_bin = 0; i_bin < NUM_BINS_(i_section); i_bin++) {
      aero_phase_idx -= NUM_PHASE_(i_section);
      if (aero_phase_idx < 0) {
        *radius = EFFECTIVE_RADIUS_(i_section, i_bin);
        i_section = NUM_SECTION_;
        break;
      }
    }
  }

  return (void *) &(float_data[FLOAT_DATA_SIZE_]);
}

/** \brief Get the particle number concentration \f$n\f$ (\f$\mbox{\si{\#\per\cubic\centi\metre}}\f$)
 *
 * The modal mass number concentration is calculated for a log-normal
 * distribution where the geometric mean diameter (\f$\tilde{D}_n\f$) and
 * geometric standard deviation (\f$\tilde{\sigma}_g\f$) are set by the aerosol
 * model prior to solving the chemistry. Thus, all
 * \f$\frac{\partial n}{\partial y}\f$ are zero. The number concentration is
 * calculated according to the equation given in Table 1 of Zender
 * \cite Zender2002 :
 * \f[
 *      n = N_0 = \frac{6V_0}{\pi}\tilde{D}_n^{-3}e^{-9\tilde{\sigma}_g^2/2}
 * \f]
 * \f[
 *      V_0 = \sum_i{\rho_im_i}
 * \f]
 * where \f$\rho_i\f$ and \f$m_i\f$ are the density and total mass of species
 * \f$i\f$ in the specified mode.
 *
 * \param aero_phase_idx Index of the aerosol phase within the representation
 * \param number_conc Particle number concentration, \f$n\f$
 *                    (\f$\mbox{\si{\#\per\cubic\centi\metre}}\f$)
 * \param partial_deriv \f$\frac{\partial n}{\partial y}\f$ where \f$y\f$ are
 *                      the species on the state array
 * \param aero_rep_data Pointer to the aerosol representation data
 * \return The aero_rep_data pointer advanced by the size of the aerosol
 *         representation
 */
__device__ void *aero_rep_gpu_modal_binned_mass_get_number_conc(int aero_phase_idx,
                                                 double *number_conc, double *partial_deriv, void *aero_rep_data) {
  int *int_data = (int *) aero_rep_data;
  double *float_data = (double *) &(int_data[INT_DATA_SIZE_]);

  for (int i_section = 0; i_section < NUM_SECTION_ && aero_phase_idx >= 0;
       i_section++) {
    for (int i_bin = 0; i_bin < NUM_BINS_(i_section) && aero_phase_idx >= 0;
         i_bin++) {
      aero_phase_idx -= NUM_PHASE_(i_section);
      if (aero_phase_idx < 0) {
        *number_conc = NUMBER_CONC_(i_section, i_bin);
        i_section = NUM_SECTION_;
        break;
      }
    }
  }

  return (void *) &(float_data[FLOAT_DATA_SIZE_]);
}

/** \brief Get the type of aerosol concentration used.
 *
 * Modal mass concentrations are per-mode or per-bin.
 *
 * \param aero_phase_idx Index of the aerosol phase within the representation
 * \param aero_conc_type Pointer to int that will hold the concentration type
 *                       code
 * \param aero_rep_data Pointer to the aerosol representation data
 * \return The aero_rep_data pointer advanced by the size of the aerosol
 *         representation
 */
__device__ void *aero_rep_gpu_modal_binned_mass_get_aero_conc_type(int aero_phase_idx,
                                                    int *aero_conc_type, void *aero_rep_data) {
  int *int_data = (int *) aero_rep_data;
  double *float_data = (double *) &(int_data[INT_DATA_SIZE_]);

  *aero_conc_type = 1;

  return (void *) &(float_data[FLOAT_DATA_SIZE_]);
}

/** \brief Get the total mass in an aerosol phase \f$m\f$ (\f$\mbox{\si{\micro\gram\per\cubic\metre}}\f$)
 *
 * \param aero_phase_idx Index of the aerosol phase within the representation
 * \param aero_phase_mass Total mass in the aerosol phase, \f$m\f$
 *                        (\f$\mbox{\si{\micro\gram\per\cubic\metre}}\f$)
 * \param aero_phase_avg_MW Average molecular weight in the aerosol phase
 *                          (\f$\mbox{\si{\kilogram\per\mole}}\f$)
 * \param partial_deriv \f$\frac{\partial m}{\partial y}\f$ where \f$y\f$ are
 *                      the species on the state array
 * \param aero_rep_data Pointer to the aerosol representation data
 * \return The aero_rep_data pointer advanced by the size of the aerosol
 *         representation
 */
__device__ void * aero_rep_gpu_modal_binned_mass_get_aero_phase_mass(int aero_phase_idx,
                                                      double *aero_phase_mass, double *aero_phase_avg_MW,
                                                      double *partial_deriv, void *aero_rep_data)
{
  int *int_data = (int*) aero_rep_data;
  double *float_data = (double*) &(int_data[INT_DATA_SIZE_]);

  for (int i_section=0; i_section<NUM_SECTION_ && aero_phase_idx>=0;
       i_section++) {
    for (int i_phase=0; i_phase<NUM_PHASE_(i_section) && aero_phase_idx>=0;
         i_phase++) {
      for (int i_bin=0; i_bin<NUM_BINS_(i_section) && aero_phase_idx>=0;
           i_bin++) {
        if (aero_phase_idx==0) {
          *aero_phase_mass = PHASE_MASS_(i_section, i_phase, i_bin);
          *aero_phase_avg_MW = PHASE_AVG_MW_(i_section, i_phase, i_bin);
        }
        aero_phase_idx-=1;
      }
    }
  }

  return (void*) &(float_data[FLOAT_DATA_SIZE_]);
}

/** \brief Advance the aerosol representation data pointer to the next aerosol representation
 *
 * \param aero_rep_data Pointer to the aerosol representation data
 * \return The aero_rep_data pointer advanced by the size of the aerosol
 *         representation data
 */
__device__ void *aero_rep_gpu_modal_binned_mass_skip(void *aero_rep_data) {
  int *int_data = (int *) aero_rep_data;
  double *float_data = (double *) &(int_data[INT_DATA_SIZE_]);

  return (void *) &(float_data[FLOAT_DATA_SIZE_]);
}


#undef BINNED
#undef MODAL

#undef TEMPERATURE_K_
#undef PRESSURE_PA_

#undef UPDATE_GSD
#undef UPDATE_GMD

#undef NUM_SECTION_
#undef INT_DATA_SIZE_
#undef FLOAT_DATA_SIZE_
#undef AERO_REP_ID_
#undef NUM_INT_PROP_
#undef NUM_FLOAT_PROP_
#undef MODE_INT_PROP_LOC_
#undef MODE_FLOAT_PROP_LOC_
#undef SECTION_TYPE_
#undef NUM_BINS_
#undef NUM_PHASE_
#undef PHASE_STATE_ID_
#undef PHASE_MODEL_DATA_ID_
#undef GMD_
#undef BIN_DP_
#undef GSD_
#undef NUMBER_CONC_
#undef EFFECTIVE_RADIUS_
#undef PHASE_MASS_
#undef PHASE_AVG_MW_

}
