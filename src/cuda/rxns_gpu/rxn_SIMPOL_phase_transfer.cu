#include "hip/hip_runtime.h"
/* Copyright (C) 2015-2018 Matthew Dawson
 * Licensed under the GNU General Public License version 2 or (at your
 * option) any later version. See the file COPYING for details.
 *
 * Phase Transfer reaction solver functions
 *
*/
/** \file
 * \brief Phase Transfer reaction solver functions
*/
extern "C"{
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "../aeros_gpu/aero_rep_solver_gpu.h"
#include "../aeros_gpu/sub_model_solver_gpu.h"

// TODO Lookup environmental indices during initialization
#define TEMPERATURE_K_ env_data[0]
#define PRESSURE_PA_ env_data[1]

// Universal gas constant (J/mol/K)
#define UNIV_GAS_CONST_ 8.314472
// Small number for ignoring low concentrations
#define VERY_SMALL_NUMBER_ 1.0e-30

#define DELTA_H_ float_data[0*n_rxn]
#define DELTA_S_ float_data[1*n_rxn]
#define DIFF_COEFF_ float_data[2*n_rxn]
#define PRE_C_AVG_ float_data[3*n_rxn]
#define B1_ float_data[4*n_rxn]
#define B2_ float_data[5*n_rxn]
#define B3_ float_data[6*n_rxn]
#define B4_ float_data[7*n_rxn]
#define C_AVG_ALHPA_ float_data[8*n_rxn]
#define EQUIL_CONST_ float_data[9*n_rxn]
#define CONV_ float_data[10*n_rxn]
#define MW_ float_data[11*n_rxn]
#define UGM3_TO_PPM_ float_data[12*n_rxn]
#define SMALL_NUMBER_ float_data[13*n_rxn]
#define NUM_AERO_PHASE_ int_data[0*n_rxn]
#define GAS_SPEC_ (int_data[1*n_rxn]-1)
#define NUM_INT_PROP_ 2
#define NUM_FLOAT_PROP_ 14
#define AERO_SPEC_(x) (int_data[(NUM_INT_PROP_ + x)*n_rxn]-1)
#define AERO_ACT_ID_(x) (int_data[(NUM_INT_PROP_ + NUM_AERO_PHASE_ + x)*n_rxn])
#define AERO_PHASE_ID_(x) (int_data[(NUM_INT_PROP_ + 2*(NUM_AERO_PHASE_) + x)*n_rxn]-1)
#define AERO_REP_ID_(x) (int_data[(NUM_INT_PROP_ + 3*(NUM_AERO_PHASE_) + x)*n_rxn]-1)
#define DERIV_ID_(x) (int_data[(NUM_INT_PROP_ + 4*(NUM_AERO_PHASE_) + x)*n_rxn])
#define JAC_ID_(x) (int_data[(NUM_INT_PROP_ + 1 + 5*(NUM_AERO_PHASE_) + x)*n_rxn])
#define FAST_FLUX_(x) (float_data[(NUM_FLOAT_PROP_ + x)*n_rxn])
#define AERO_ADJ_(x) (float_data[(NUM_FLOAT_PROP_ + NUM_AERO_PHASE_ + x)*n_rxn])
#define INT_DATA_SIZE_ (NUM_INT_PROP_+2+8*(NUM_AERO_PHASE_))
#define FLOAT_DATA_SIZE_ (NUM_FLOAT_PROP_+2*(NUM_AERO_PHASE_))


/** \brief Update reaction data for new environmental conditions
 *
 * For Phase Transfer reaction this only involves recalculating the rate
 * constant.
 *
 * \param env_data Pointer to the environmental state array
 * \param rxn_data Pointer to the reaction data
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
__device__ void rxn_gpu_SIMPOL_phase_transfer_update_env_state(double *rate_constants,
   int n_rxn2,double *double_pointer_gpu, double *env_data,
                                                  void *rxn_data)
{
  int n_rxn=n_rxn2;
  int *int_data = (int*) rxn_data;
  double *float_data = double_pointer_gpu;

  // Calculate the mass accomodation coefficient if the N* parameter
  // was provided, otherwise set it to 1.0
  double mass_acc = 1.0;
  if (DELTA_H_!=0.0 || DELTA_S_!=0.0) {
    double del_G = DELTA_H_ - TEMPERATURE_K_ * DELTA_S_;
    mass_acc = exp(-del_G/(UNIV_GAS_CONST_ * TEMPERATURE_K_));
    mass_acc = mass_acc / (1.0 + mass_acc);
  }

  // Save c_rms * mass_acc for use in mass transfer rate calc
  C_AVG_ALHPA_ = PRE_C_AVG_ * sqrt(TEMPERATURE_K_) * mass_acc;

  // SIMPOL.1 vapor pressure (Pa)
  double vp = B1_ / TEMPERATURE_K_
              + B2_ + B3_ * TEMPERATURE_K_
              + B4_ * log(TEMPERATURE_K_);
  vp = 101325.0 * pow(10, vp);

  // Calculate the conversion from ug_x/m^3 -> ppm_x
  UGM3_TO_PPM_ = CONV_ * TEMPERATURE_K_ / PRESSURE_PA_;

  // Calculate the partitioning coefficient K_eq (ppm_x/ug_x*ug_tot/kg_tot)
  // such that for partitioning species X at equilibrium:
  //   [X]_gas = [X]_aero * activity_coeff_X * K_eq * MW_tot_aero / [tot]_aero
  // where 'tot' indicates all species within an aerosol phase combined
  // with []_gas in (ppm) and []_aero in (ug/m^3)
  EQUIL_CONST_ = vp                    // (Pa_x*mol_tot/mol_x)
                 / PRESSURE_PA_       // (1/Pa_air)
                 / MW_                // (mol_x/kg_x)
                 * 1.0e6;             // 1.0e6ppm_x*Pa_air/Pa_x *
  //  1.0e-9kg_x/ug_x * 1.0e9ug_tot/kg_tot

}



/** \brief Calculate contributions to the time derivative \f$f(t,y)\f$ from
 * this reaction.
 *
 * \param model_data Pointer to the model data, including the state array
 * \param deriv Pointer to the time derivative to add contributions to
 * \param rxn_data Pointer to the reaction data
 * \param time_step Current time step being computed (s)
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
#ifdef PMC_USE_SUNDIALS
__device__ void rxn_gpu_SIMPOL_phase_transfer_calc_deriv_contrib(double *rate_constants, double *state,
          double *deriv, void *rxn_data, double * double_pointer_gpu, double time_step, int n_rxn2)
{
  int n_rxn=n_rxn2;
  int *int_data = (int*) rxn_data;
  double *float_data = double_pointer_gpu;

  /*
  // Calculate derivative contributions for each aerosol phase
  for (int i_phase=0; i_phase<NUM_AERO_PHASE_; i_phase++) {

    // Skip reactions that are being treated as instantaneous
    if (FAST_FLUX_(i_phase) != 0.0) continue;

    // Get the particle effective radius (m)
    double radius;
    aero_rep_gpu_get_effective_radius(
		  model_data,			// model data
		  AERO_REP_ID_(i_phase),	// aerosol representation index
		  AERO_PHASE_ID_(i_phase),	// aerosol phase index
		  &radius);			// particle effective radius (m)

    // Get the particle number concentration (#/cc)
    double number_conc;
    aero_rep_gpu_get_number_conc(
		  model_data,			// model data
		  AERO_REP_ID_(i_phase),	// aerosol representation index
		  AERO_PHASE_ID_(i_phase),	// aerosol phase index
		  &number_conc);		// particle number conc (#/cc)

    // Check the aerosol concentration type (per-particle or total per-phase
    // mass)
    int aero_conc_type = aero_rep_gpu_get_aero_conc_type(
		  model_data,			// model data
		  AERO_REP_ID_(i_phase),	// aerosol representation index
		  AERO_PHASE_ID_(i_phase));	// aerosol phase index

    // Get the total mass of the aerosol phase
    double aero_phase_gpu_mass;
    double aero_phase_gpu_avg_MW;
    aero_rep_gpu_get_aero_phase_mass(
                  model_data,                   // model data
                  AERO_REP_ID_(i_phase),        // aerosol representation index
                  AERO_PHASE_ID_(i_phase),      // aerosol phase index
                  &aero_phase_gpu_mass,             // total aerosol-phase mass
                  &aero_phase_gpu_avg_MW);          // avg MW in the aerosol phase

    // If the radius, number concentration, or aerosol-phase mass are zero,
    // no transfer occurs
    if (radius <= ZERO || number_conc <= ZERO || aero_phase_gpu_mass <= ZERO) continue;

    // Calculate the rate constant for diffusion limited mass transfer to the
    // aerosol phase (1/s)
    double cond_rate = number_conc/(radius*radius/(3.0*DIFF_COEFF_) +
              4.0*radius/(3.0*C_AVG_ALHPA_));

    // Calculate the evaporation rate constant (ppm_x*m^3/ug_x/s)
    double evap_rate = cond_rate * (EQUIL_CONST_ * aero_phase_gpu_avg_MW /
              aero_phase_gpu_mass);

    // Slow down condensation rate as gas-phase concentrations become small
    double gas_adj = state[GAS_SPEC_] - VERY_SMALL_NUMBER_;
    gas_adj = ( gas_adj > ZERO ) ? gas_adj : ZERO;
    double cond_scaling =
      2.0 / ( 1.0 + exp( -gas_adj / SMALL_NUMBER_ ) ) - 1.0;
    cond_scaling *= cond_scaling;

    // Calculate gas-phase condensation rate (ppm/s)
    cond_rate *= state[GAS_SPEC_] * cond_scaling;

    // Get the activity coefficient (if one exists)
    double act_coeff = 1.0;
    if (AERO_ACT_ID_(i_phase)>-1) {
      act_coeff = sub_model_gpu_get_parameter_value(model_data,
                AERO_ACT_ID_(i_phase));
    }

    // Slow down evaporation as aerosol-phase activity becomes small
    double aero_adj = state[AERO_SPEC_(i_phase)] * act_coeff -
                        VERY_SMALL_NUMBER_;
    aero_adj = ( aero_adj > ZERO ) ? aero_adj : ZERO;
    double evap_scaling =
      2.0 / ( 1.0 + exp( -aero_adj / SMALL_NUMBER_ ) ) - 1.0;
    evap_scaling *= evap_scaling;

    // Calculate aerosol-phase evaporation rate (ppm/s)
    // (Slow down evaporation as aerosol-phase concentrations approach zero
    //  to help out the solver.)
    evap_rate *= state[AERO_SPEC_(i_phase)] * act_coeff * evap_scaling;

    // Change in the gas-phase is evaporation - condensation (ppm/s)
    //if (DERIV_ID_(0)>=0) deriv[DERIV_ID_(0)] += evap_rate - cond_rate;
    if (DERIV_ID_(0)>=0) atomicAdd((double*)&(deriv[DERIV_ID_(0)]),evap_rate - cond_rate);

    // Change in the aerosol-phase species is condensation - evaporation
    // (ug/m^3/s)
    if (DERIV_ID_(1+i_phase)>=0) {
      if (aero_conc_type==0) {
        // Per-particle condensation
        //deriv[DERIV_ID_(1+i_phase)] += (cond_rate - evap_rate) /
        //        UGM3_TO_PPM_ / number_conc;
        atomicAdd((double*)&(deriv[DERIV_ID_(1+i_phase)]),(cond_rate - evap_rate) /
          UGM3_TO_PPM_ / number_conc);

      } else {
        // Total aerosol mass condensation
        //deriv[DERIV_ID_(1+i_phase)] += (cond_rate - evap_rate) /
        //        UGM3_TO_PPM_;
        atomicAdd((double*)&(deriv[DERIV_ID_(1+i_phase)]),(cond_rate - evap_rate) /
          UGM3_TO_PPM_);
      }
    }
  }
*/

}
#endif


/** \brief Calculate contributions to the time derivative \f$f(t,y)\f$ from
 * this reaction.
 *
 * \param model_data Pointer to the model data, including the state array
 * \param deriv Pointer to the time derivative to add contributions to
 * \param rxn_data Pointer to the reaction data
 * \param time_step Current time step being computed (s)
 * \return The rxn_data pointer advanced by the size of the reaction data
 */

/** \brief Calculate contributions to the Jacobian from this reaction
 *
 * \param model_data Pointer to the model data
 * \param J Pointer to the sparse Jacobian matrix to add contributions to
 * \param rxn_data Pointer to the reaction data
 * \param time_step Current time step being calculated (s)
 * \return The rxn_data pointer advanced by the size of the reaction data
 */

#ifdef PMC_USE_SUNDIALS
__device__ void rxn_gpu_SIMPOL_phase_transfer_calc_jac_contrib(double *rate_constants, double *state,
          double *J, void *rxn_data, double * double_pointer_gpu, double time_step, int n_rxn2)
{
  int n_rxn=n_rxn2;
  int *int_data = (int*) rxn_data;
  double *float_data = double_pointer_gpu;

  /*
  // Calculate derivative contributions for each aerosol phase
  for (int i_phase=0; i_phase<NUM_AERO_PHASE_; i_phase++) {

    // Skip reactions that are being treated as instantaneous
    if (FAST_FLUX_(i_phase) != 0.0) continue;

    // Get the particle effective radius (m)
    double radius;
    aero_rep_gpu_get_effective_radius(
		  model_data,			// model data
		  AERO_REP_ID_(i_phase),	// aerosol representation index
		  AERO_PHASE_ID_(i_phase),	// aerosol phase index
		  &radius);			// particle effective radius (m)

    // Get the particle number concentration (#/cc)
    double number_conc;
    aero_rep_gpu_get_number_conc(
		  model_data,			// model data
		  AERO_REP_ID_(i_phase),	// aerosol representation index
		  AERO_PHASE_ID_(i_phase),	// aerosol phase index
		  &number_conc);		// particle number conc (#/cc)

    // Check the aerosol concentration type (per-particle or total per-phase mass)
    int aero_conc_type = aero_rep_gpu_get_aero_conc_type(
		  model_data,			// model data
		  AERO_REP_ID_(i_phase),	// aerosol representation index
		  AERO_PHASE_ID_(i_phase));	// aerosol phase index

    // Get the total mass of the aerosol phase
    double aero_phase_gpu_mass;
    double aero_phase_gpu_avg_MW;
    aero_rep_gpu_get_aero_phase_mass(
                  model_data,                   // model data
                  AERO_REP_ID_(i_phase),       // aerosol representation index
                  AERO_PHASE_ID_(i_phase),     // aerosol phase index
                  &aero_phase_gpu_mass,             // total aerosol-phase mass
                  &aero_phase_gpu_avg_MW);          // avg MW in the aerosol phase

    // If the radius, number concentration, or aerosol-phase mass are zero,
    // no transfer occurs
    if (radius <= ZERO || number_conc <= ZERO || aero_phase_gpu_mass <= ZERO) continue;

    // Calculate the rate constant for diffusion limited mass transfer to the
    // aerosol phase (1/s)
    double cond_rate = number_conc/(radius*radius/(3.0*DIFF_COEFF_) +
              4.0*radius/(3.0*C_AVG_ALHPA_));

    // Calculate the evaporation rate constant (ppm_x*m^3/ug_x/s)
    double evap_rate = cond_rate * (EQUIL_CONST_ * aero_phase_gpu_avg_MW /
              aero_phase_gpu_mass);

    // Slow down condensation rate as gas-phase concentrations become small
    double gas_adj = state[GAS_SPEC_] - VERY_SMALL_NUMBER_;
    gas_adj = ( gas_adj > ZERO ) ? gas_adj : ZERO;
    double cond_scaling =
      2.0 / ( 1.0 + exp( -gas_adj / SMALL_NUMBER_ ) ) - 1.0;
    double cond_scaling_deriv =
      2.0 / ( SMALL_NUMBER_ * ( exp(  gas_adj / SMALL_NUMBER_ ) + 2.0 +
                                exp( -gas_adj / SMALL_NUMBER_ ) ) );
    cond_scaling_deriv *= 2.0 * cond_scaling;
    cond_scaling *= cond_scaling;

    // Get the activity coefficient (if one exists)
    double act_coeff = 1.0;
    if (AERO_ACT_ID_(i_phase)>-1) {
      act_coeff = sub_model_gpu_get_parameter_value(model_data,
                AERO_ACT_ID_(i_phase));
    }

    // Slow down evaporation as aerosol-phase activity becomes small
    double aero_adj = state[AERO_SPEC_(i_phase)] * act_coeff -
                        VERY_SMALL_NUMBER_;
    aero_adj = ( aero_adj > ZERO ) ? aero_adj : ZERO;
    double evap_scaling =
      2.0 / ( 1.0 + exp( -aero_adj / SMALL_NUMBER_ ) ) - 1.0;
    double evap_scaling_deriv =
      2.0 / ( SMALL_NUMBER_ * ( exp(  aero_adj / SMALL_NUMBER_ ) + 2.0 +
                                exp( -aero_adj / SMALL_NUMBER_ ) ) );
    evap_scaling_deriv *= 2.0 * evap_scaling;
    evap_scaling *= evap_scaling;

    // Change in the gas-phase is evaporation - condensation (ppm/s)
      if (JAC_ID_(1+i_phase*3+1)>=0)
          J[JAC_ID_(1+i_phase*3+1)] += evap_rate * act_coeff *
                                       ( evap_scaling +
                                         state[AERO_SPEC_(i_phase)] *
                                         evap_scaling_deriv );
      if (JAC_ID_(0)>=0) J[JAC_ID_(0)] -= cond_rate *
                                          ( cond_scaling +
                                            state[GAS_SPEC_] *
                                            cond_scaling_deriv );

    // Change in the aerosol-phase species is condensation - evaporation
    // (ug/m^3/s)
    if (aero_conc_type==0) {
      // Per-particle condensation
      if (JAC_ID_(1+i_phase*3)>=0) J[JAC_ID_(1+i_phase*3)] +=
          cond_rate / number_conc / UGM3_TO_PPM_ *
          ( cond_scaling + state[GAS_SPEC_] * cond_scaling_deriv );
      if (JAC_ID_(1+i_phase*3+2)>=0) J[JAC_ID_(1+i_phase*3+2)] -=
          evap_rate * act_coeff / number_conc / UGM3_TO_PPM_ *
          ( evap_scaling + state[AERO_SPEC_(i_phase)] * evap_scaling_deriv );
    } else {
      // Total aerosol mass condensation
      if (JAC_ID_(1+i_phase*3)>=0) J[JAC_ID_(1+i_phase*3)] +=
          cond_rate / UGM3_TO_PPM_ *
          ( cond_scaling + state[GAS_SPEC_] * cond_scaling_deriv );
      if (JAC_ID_(1+i_phase*3+2)>=0) J[JAC_ID_(1+i_phase*3+2)] -=
          evap_rate * act_coeff / UGM3_TO_PPM_ *
          ( evap_scaling + state[AERO_SPEC_(i_phase)] * evap_scaling_deriv );
    }

  }
*/

}
#endif

/** \brief Retrieve Int data size
 *
 * \param rxn_data Pointer to the reaction data
 * \return The data size of int array
 */
void * rxn_gpu_SIMPOL_phase_transfer_int_size(void *rxn_data)
{
  int n_rxn=1;
  int *int_data = (int*) rxn_data;
  double *float_data = (double*) &(int_data[INT_DATA_SIZE_]);

  return (void*) float_data;
}

/** \brief Advance the reaction data pointer to the next reaction
 *
 * \param rxn_data Pointer to the reaction data
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
void * rxn_gpu_SIMPOL_phase_transfer_skip(void *rxn_data)
{
  int n_rxn=1;
  int *int_data = (int*) rxn_data;
  double *float_data = (double*) &(int_data[INT_DATA_SIZE_]);

  return (void*) &(float_data[FLOAT_DATA_SIZE_]);
}

/** \brief Print the Phase Transfer reaction parameters
 *
 * \param rxn_data Pointer to the reaction data
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
void * rxn_gpu_SIMPOL_phase_transfer_print(void *rxn_data)
{
  int n_rxn=1;
  int *int_data = (int*) rxn_data;
  double *float_data = (double*) &(int_data[INT_DATA_SIZE_]);

  printf("\n\nSIMPOL.1 Phase Transfer reaction\n");
  for (int i=0; i<INT_DATA_SIZE_; i++)
    printf("  int param %d = %d\n", i, int_data[i]);
  for (int i=0; i<FLOAT_DATA_SIZE_; i++)
    printf("  float param %d = %le\n", i, float_data[i]);

  return (void*) &(float_data[FLOAT_DATA_SIZE_]);
}


#undef TEMPERATURE_K_
#undef PRESSURE_PA_

#undef UNIV_GAS_CONST_
#undef VERY_SMALL_NUMBER_
#undef RATE_SCALING_

#undef DELTA_H_
#undef DELTA_S_
#undef DIFF_COEFF_
#undef PRE_C_AVG_
#undef B1_
#undef B2_
#undef B3_
#undef B4_
#undef C_AVG_ALHPA_
#undef EQUIL_CONST_
#undef CONV_
#undef MW_
#undef UGM3_TO_PPM_
#undef SMALL_NUMBER_
#undef NUM_AERO_PHASE_
#undef GAS_SPEC_
#undef NUM_INT_PROP_
#undef NUM_FLOAT_PROP_
#undef AERO_SPEC_
#undef AERO_ACT_ID_
#undef AERO_PHASE_ID_
#undef AERO_REP_ID_
#undef DERIV_ID_
#undef JAC_ID_
#undef FAST_FLUX_
#undef AERO_ADJ_
#undef INT_DATA_SIZE_
#undef FLOAT_DATA_SIZE_
}