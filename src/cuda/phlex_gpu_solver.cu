#include "hip/hip_runtime.h"
//Test
extern "C" {
#include "phlex_gpu_solver.h"
#include "rxns_gpu.h"
//#include "phlex_solver.h"
//}

// Reaction types (Must match parameters defined in pmc_rxn_factory)
#define RXN_ARRHENIUS 1
#define RXN_TROE 2
#define RXN_CMAQ_H2O2 3
#define RXN_CMAQ_OH_HNO3 4
#define RXN_PHOTOLYSIS 5
#define RXN_HL_PHASE_TRANSFER 6
#define RXN_AQUEOUS_EQUILIBRIUM 7
#define RXN_ZSR_AEROSOL_WATER 8
#define RXN_PDFITE_ACTIVITY 9
#define RXN_SIMPOL_PHASE_TRANSFER 10
#define RXN_CONDENSED_PHASE_ARRHENIUS 11
#define RXN_FIRST_ORDER_LOSS 12
#define RXN_EMISSION 13
#define RXN_WET_DEPOSITION 14

#define CHEM_SPEC_VARIABLE 1

//TtODO: change doubles per float (since max tolerance is only at E-12)

ModelDatagpu *mdgpu;
ModelDatagpu *mdcpu;
double *derivgpu_data;
double *deriv_cpu;
size_t deriv_size;
unsigned int countergpu = 0;

size_t state_size;
size_t env_size;
size_t rate_constants_size;
bool solver_set_gpu_sizes = 1;
int *int_pointer;
double *double_pointer;
int *int_pointer_gpu;
double *double_pointer_gpu;
unsigned int *start_rxn_param;
unsigned int *dev_start_rxn_param;
unsigned int int_max_size = 0;
unsigned int double_max_size = 0;
double *state_gpu;
double *state_cpu;
double *env_gpu;
double *rate_constants_gpu;
double *rate_constants_cpu;

//unsigned int *int_sizes;
//unsigned int *double_sizes;

static void HandleError(hipError_t err,
                        const char *file,
                        int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err),
           file, line);
    exit(EXIT_FAILURE);
  }
}

static void HandleError2(const char *file,
                         int line) {
  hipError_t err;
  err=hipGetLastError();
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err),
           file, line);    exit(EXIT_FAILURE);
  }
}

void solver_new_gpu_cu(SolverDatagpu *sd, int n_dep_var,
                       int n_state_var, int *var_type, int n_rxn,
                       int n_rxn_int_param, int n_rxn_float_param,
                       int n_cells_aux) { //Ttodo: not necessary pass this parameters, there are on md

  ModelDatagpu *md = &sd->model_data;

  //Sizes
  size_t start_size = (n_rxn+1) * sizeof(unsigned int);
  state_size = n_state_var*n_cells_aux * sizeof(double); //TODO: Adapt state to has the same size as deriv?
  deriv_size = n_dep_var*n_cells_aux * sizeof(sd->y);
  env_size = 2*n_cells_aux * sizeof(double); //Temp and pressure
  rate_constants_size = n_rxn * n_cells_aux * sizeof(double);

  printf("n_rxn: %d " , n_rxn);
  //printf("n_rxn_float_param: %d ", n_rxn_float_param);
  //printf("n_rxn_int_param: %d ", n_rxn_int_param);
  printf("n_state_var: %d" ,n_state_var);
  printf("n_dep_var: %d" ,n_dep_var);

  //Create started indexes of arrays
  start_rxn_param = (unsigned int *) malloc(start_size);

  rate_constants_cpu = (double *) malloc(rate_constants_size);


  //GPU allocation
  hipMalloc((void **) &dev_start_rxn_param, start_size);

  //realtype *deriv_data = N_VGetArrayPointer(sd->y);
  //HANDLE_ERROR(hipHostRegister(deriv_data, deriv_size, hipHostRegisterPortable));//pinned

  hipMalloc((void **) &derivgpu_data, deriv_size);
  hipMalloc((void **) &state_gpu, state_size);
  hipMalloc((void **) &env_gpu, env_size);
  hipMalloc((void **) &rate_constants_gpu, rate_constants_size);


  //HANDLE_ERROR(hipHostRegister(deriv_data, deriv_size, hipHostRegisterMapped));//pinned, not work properly
  //HANDLE_ERROR(hipHostRegister(deriv_data, deriv_size, hipHostRegisterDefault));
  //HANDLE_ERROR(hipHostGetDevicePointer((void**) &(derivgpu_data), (void*)deriv_data, 0));

  hipHostMalloc((void**)&deriv_cpu, deriv_size);//pinned
  //hipHostMalloc((void**)&state_cpu, state_size);//pinned

  //hipMalloc((void **) &mdgpu, sizeof(*mdgpu));

  //HANDLE_ERROR(hipHostGetDevicePointer((void**) &(derivgpu_data), (void*)deriv_data, 0));

  if (deriv_size/(sizeof(double)) > MAX_SHARED_MEMORY_BLOCK_DOUBLE)
#ifndef FAILURE_DETAIL
    printf("\nMore solver variables(deriv): %d than maximum shared memory: %d",
           deriv_size, MAX_SHARED_MEMORY_BLOCK_DOUBLE);
#endif

}

void solver_update_state_gpu(ModelDatagpu *md) {//HANDLE_ERROR(hipMemcpy(mdgpu->state, md->state, state_size*sizeof(int), hipMemcpyHostToDevice));
}

void solver_set_data_gpu(ModelDatagpu *model_data) {
  //Get rxn sizes
  if (solver_set_gpu_sizes) {

    //HANDLE_ERROR(hipHostRegister(model_data->state, state_size, hipHostRegisterPortable)); pinned

    // Get the number of reactions
    int *rxn_data = (int *) (model_data->rxn_data);
    int n_rxn = *(rxn_data++);
    void *rxn_param = (void *) rxn_data;
    int *float_data = (int *) rxn_data;
    unsigned int int_size = 0;
    unsigned int int_total_size = 0;
    unsigned int double_size = 0;
    unsigned int double_total_size = 0;

    size_t start_size = (n_rxn+1) * sizeof(unsigned int);
    unsigned int int_sizes[start_size];
    unsigned int double_sizes[start_size];

    for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {

      //Reaction distances between pointers rows
      start_rxn_param[i_rxn] = (unsigned int) ((int *) rxn_data - (int *) rxn_param);

      int *rxn_start = rxn_data;

      // Get the reaction type
      int rxn_type = *(rxn_data++);

      // Call the appropriate function
      switch (rxn_type) {
        case RXN_AQUEOUS_EQUILIBRIUM :
          float_data = (int *) rxn_gpu_aqueous_equilibrium_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_aqueous_equilibrium_skip((void *) rxn_data);
          break;
        case RXN_ARRHENIUS :
          float_data = (int *) rxn_gpu_arrhenius_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_arrhenius_skip((void *) rxn_data);
          break;
        case RXN_CMAQ_H2O2 :
          float_data = (int *) rxn_gpu_CMAQ_H2O2_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_CMAQ_H2O2_skip((void *) rxn_data);
          break;
        case RXN_CMAQ_OH_HNO3 :
          float_data = (int *) rxn_gpu_CMAQ_OH_HNO3_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_CMAQ_OH_HNO3_skip((void *) rxn_data);
          break;
        case RXN_CONDENSED_PHASE_ARRHENIUS :
          float_data = (int *) rxn_gpu_condensed_phase_arrhenius_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_condensed_phase_arrhenius_skip((void *) rxn_data);
          break;
        case RXN_EMISSION :
          float_data = (int *) rxn_gpu_emission_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_emission_skip((void *) rxn_data);
          break;
        case RXN_FIRST_ORDER_LOSS :
          float_data = (int *) rxn_gpu_first_order_loss_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_first_order_loss_skip((void *) rxn_data);
          break;
        case RXN_HL_PHASE_TRANSFER :
          float_data = (int*) rxn_gpu_HL_phase_transfer_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_HL_phase_transfer_skip((void *) rxn_data);
          break;
        case RXN_PDFITE_ACTIVITY :
          float_data = (int *) rxn_gpu_PDFiTE_activity_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_PDFiTE_activity_skip((void *) rxn_data);
          break;
        case RXN_PHOTOLYSIS :
          float_data = (int *) rxn_gpu_photolysis_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_photolysis_skip((void *) rxn_data);
          break;
        case RXN_SIMPOL_PHASE_TRANSFER :
          float_data = (int *) rxn_gpu_SIMPOL_phase_transfer_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_SIMPOL_phase_transfer_skip((void *) rxn_data);
          break;
        case RXN_TROE :
          float_data = (int *) rxn_gpu_troe_int_size(
                  (void *) rxn_data);
          rxn_data =(int*)rxn_gpu_troe_skip((void *) rxn_data);
          break;
        case RXN_WET_DEPOSITION :
          float_data = (int *) rxn_gpu_wet_deposition_int_size(
                  (void *) rxn_data);
          rxn_data =(int*)rxn_gpu_wet_deposition_skip((void *) rxn_data);
          break;
        case RXN_ZSR_AEROSOL_WATER :
          float_data = (int *) rxn_gpu_ZSR_aerosol_water_int_size(
                  (void *) rxn_data);
          rxn_data = (int*) rxn_gpu_ZSR_aerosol_water_skip((void *) rxn_data);
          break;
      }

      int_size = (unsigned int) ((int *) float_data - (int *) rxn_start);
      int_total_size += int_size;
      int_sizes[i_rxn+1] = int_total_size;
      if(int_size>int_max_size) int_max_size=int_size;

      double_size = (unsigned int) ((double *) rxn_data - (double *) float_data);
      double_total_size += double_size;
      double_sizes[i_rxn+1] = double_total_size;
      if(double_size>double_max_size) double_max_size=double_size;

    }

    //Ttodo: best option is put sizes array of rxn insdie rxn matrix taking advantage of read memory, or avoid zeros
    int_sizes[0]=0;
    double_sizes[0]=0;
    unsigned int rxn_int_size=n_rxn*int_max_size;
    unsigned int rxn_double_size=n_rxn*double_max_size;

    //Allocate int and double rxn data separately
    int_pointer = (int *) malloc(rxn_int_size * sizeof(int));
    memset(int_pointer, -1, rxn_int_size * sizeof(int));
    double_pointer = (double*)calloc(rxn_double_size, sizeof(double));
    hipMalloc((void **) &int_pointer_gpu, rxn_int_size * sizeof(int));
    hipMalloc((void **) &double_pointer_gpu, rxn_double_size * sizeof(double));

    //Copy into gpu rxn data
    //Rxn matrix is rotated to improve memory acces on gpu
    for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {

      int_size = int_sizes[i_rxn+1] - int_sizes[i_rxn];
      double_size = double_sizes[i_rxn+1] - double_sizes[i_rxn];

      for (int j = 0; j < int_size; j++)
        int_pointer[n_rxn*j+i_rxn] = ((int *) rxn_param)[start_rxn_param[i_rxn] + j]; //[int_size][n_rxn]

      for (int j = 0; j < double_size; j++) {
        double *float_data = (double *) &(((int *) rxn_param)[start_rxn_param[i_rxn] + int_size]);
        double_pointer[n_rxn*j+i_rxn] = float_data[j];//[int_size][n_rxn]
      }

    }
    //Ttodo: Quick sort to reorganize rows starting with low number of zeros to a lot of zeros in row

    printf(" Zeros_added_int_%: %f ", ((double) (n_rxn*int_max_size))/(n_rxn*int_max_size-int_total_size));
    printf(" Zeros_added_double_%: %f\n ", ((double) (n_rxn*double_max_size))/(n_rxn*double_max_size-double_total_size));

    HANDLE_ERROR(hipMemcpy(int_pointer_gpu, int_pointer, rxn_int_size*sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(double_pointer_gpu, double_pointer, rxn_double_size*sizeof(double), hipMemcpyHostToDevice));

    //TODO: Update Some rxn values, changes on monarch each iteration(temperature/pressure-update function) by update functions
    solver_set_gpu_sizes = 0;
  }
}

__global__ void updateEnvRxnBlock(double *rate_constants2, int n_rxn_total_threads,
       int n_cells_gpu, int *int_pointer, double *env2, double *double_pointer){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  //__shared__ double rate_constants[MAX_SHARED_MEMORY_BLOCK_DOUBLE];

  if (index < n_rxn_total_threads) {

    double *rate_constants;
    double *env;
    int env_size_cell = 2; //Temperature and pressure
    int n_rxn = n_rxn_total_threads / n_cells_gpu;
    int cell = index / n_rxn;

    int *int_data = (int *) &(((int *) int_pointer)[index % n_rxn]);
    double *float_data = (double *) &(((double *) double_pointer)[index % n_rxn]);

    int rxn_type = int_data[0];
    int *rxn_data = (int *) &(int_data[1 * n_rxn]);

    env = (double *) &(((double *) env2)[env_size_cell * cell]);
    rate_constants = rate_constants2+index;

    switch (rxn_type) {
      case RXN_AQUEOUS_EQUILIBRIUM :
        //rxn_gpu_aqueous_equilibrium_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_ARRHENIUS :
        rxn_gpu_arrhenius_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_CMAQ_H2O2 :
        rxn_gpu_CMAQ_H2O2_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_CMAQ_OH_HNO3 :
        rxn_gpu_CMAQ_OH_HNO3_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_CONDENSED_PHASE_ARRHENIUS :
        rxn_gpu_condensed_phase_arrhenius_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_EMISSION :
        rxn_gpu_emission_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_FIRST_ORDER_LOSS :
        rxn_gpu_first_order_loss_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_HL_PHASE_TRANSFER :
        //rxn_gpu_HL_phase_transfer_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_PDFITE_ACTIVITY :
        rxn_gpu_PDFiTE_activity_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_PHOTOLYSIS :
        rxn_gpu_photolysis_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_SIMPOL_PHASE_TRANSFER :
        //rxn_gpu_SIMPOL_phase_transfer_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_TROE :
        rxn_gpu_troe_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_WET_DEPOSITION :
        rxn_gpu_wet_deposition_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
      case RXN_ZSR_AEROSOL_WATER :
        rxn_gpu_ZSR_aerosol_water_update_env_state(rate_constants, n_rxn, float_data, env, int_data);
        break;
    }

  }

}

void rxn_update_env_state_gpu(ModelDatagpu *model_data, double *env){

  int n_cells = model_data->n_cells;
  int *rxn_data = (int *) (model_data->rxn_data);
  int n_rxn_total_threads = rxn_data[0]*n_cells; //rxn_data[0] is rxn size
  double *rate_constants = model_data->rate_constants;

  //ttodo: Make memcpy outside work fine

  HANDLE_ERROR(hipMemcpy(env_gpu, env, env_size, hipMemcpyHostToDevice));

  //env_gpu=env;//This slow a lot calc_deriv_gpu
  //rate_constants_gpu=rate_constants;

  updateEnvRxnBlock << < (n_rxn_total_threads + MAX_N_GPU_THREAD - 1) / MAX_N_GPU_THREAD, MAX_N_GPU_THREAD >> >
  (rate_constants_gpu,n_rxn_total_threads, n_cells, int_pointer_gpu, env_gpu, double_pointer_gpu );

  HANDLE_ERROR(hipMemcpy(model_data->rate_constants, rate_constants_gpu, rate_constants_size, hipMemcpyDeviceToHost)); //this give error

  //memcpy(model_data->rate_constants, rate_constants_cpu, rate_constants_size);

}

//TODO: FIX THIS BUG (GUILLERMO OR SOMEONE) (this works, but if move this into another file with maxthreads=1024, it crash
__device__ void rxn_gpu_tmp_arrhenius2(
      ModelDatagpu *model_data,
      double *deriv, int *rxn_data, double * double_pointer_gpu,
      double time_step, int n_rxn)
{

  double *state = model_data->state;
  int *int_data = (int*) rxn_data;
  double *float_data = double_pointer_gpu;

  // Calculate the reaction rate
  double rate = float_data[6*n_rxn];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++) rate *= state[int_data[(2 + i_spec)*n_rxn]-1];

  // Add contributions to the time derivative
  if (rate!=ZERO) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0] + int_data[1*n_rxn] + i_dep_var)*n_rxn] < 0) continue;
      //deriv[DERIV_ID_(i_dep_var)] -= rate;
      atomicAdd(&(deriv[int_data[(2 + int_data[0] + int_data[1*n_rxn] + i_dep_var)*n_rxn]]),-rate);
    }
    for (int i_spec=0; i_spec<int_data[1*n_rxn]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0] + int_data[1*n_rxn] + i_dep_var)*n_rxn] < 0) continue;

      // Negative yields are allowed, but prevented from causing negative
      // concentrations that lead to solver failures
      if (-rate*float_data[(7 + i_spec)*n_rxn]*time_step <=
          state[int_data[(2 + int_data[0] + i_spec)*n_rxn]-1]) {
        //deriv[DERIV_ID_(i_dep_var)] += rate*YIELD_(i_spec);
        atomicAdd(&(deriv[int_data[(2 + int_data[0] + int_data[1*n_rxn] + i_dep_var)*n_rxn]]),
                  rate*float_data[(7 + i_spec)*n_rxn]);
      }
    }
  }
}

__global__ void solveRxnBlock(ModelDatagpu *model_data, double *state, double *deriv,
          double time_step, int deriv_length, int state_size, int n_rxn_total_threads,
          int n_cells_gpu, int *int_pointer, double *double_pointer,
          double *rate_constants2) //Interface CPU/GPU
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

   //rxn_gpu_tmp_arrhenius(
          //model_data, deriv, int_pointer, double_pointer, time_step, n_rxn
     //     );

  __shared__ double deriv_data2[MAX_SHARED_MEMORY_BLOCK_DOUBLE];

  if (threadIdx.x < deriv_length){ //This produces seg.fault for some large values seems
    deriv_data2[index] = 0.0;
  }

  //if(index==2)//!!DON'T DELETE, it gives bad result but allows take measurements for large input values
  //for (int i=0; i<deriv_length; i++)
  //deriv_data[i]=0.0;

  //if (threadIdx.x < deriv_length)//dont work?
    //for (int i_spec = threadIdx.x; i_spec < deriv_length; i_spec += blockDim.x) {
      //deriv_data[index] = 0.0; }
  __syncthreads();

  if (index < n_rxn_total_threads) {

    int state_size_cell = state_size/n_cells_gpu;

    int deriv_length_cell = deriv_length/n_cells_gpu;
    int n_rxn = n_rxn_total_threads/n_cells_gpu;
    int cell=index/n_rxn;

    int *int_data = (int *) &(((int *) int_pointer)[index%n_rxn]);
    double *float_data = (double *) &(((double *) double_pointer)[index%n_rxn]);

    int rxn_type = int_data[0];
    int *rxn_data = (int *) &(int_data[1*n_rxn]);

    double *deriv_data = &( deriv_data2[deriv_length_cell*cell]);
    //state= state+(deriv_length_cell*cell); //TODO: Fix this different size with deriv?
    state= state+(state_size_cell*cell);
    double *rate_constants = &( rate_constants2[index]);

    switch (rxn_type) {
      case RXN_AQUEOUS_EQUILIBRIUM :
        //rxn_gpu_aqueous_equilibrium_calc_deriv_contrib(rate_constants,
        //        state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_ARRHENIUS :
        rxn_gpu_arrhenius_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_CMAQ_H2O2 :
        rxn_gpu_CMAQ_H2O2_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_CMAQ_OH_HNO3 :
        rxn_gpu_CMAQ_OH_HNO3_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_CONDENSED_PHASE_ARRHENIUS :
        rxn_gpu_condensed_phase_arrhenius_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_EMISSION :
        rxn_gpu_emission_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_FIRST_ORDER_LOSS :
        rxn_gpu_first_order_loss_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_HL_PHASE_TRANSFER :
        //rxn_gpu_HL_phase_transfer_calc_deriv_contrib(rate_constants,
        //        state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_PDFITE_ACTIVITY :
        rxn_gpu_PDFiTE_activity_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_PHOTOLYSIS :
        rxn_gpu_photolysis_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_SIMPOL_PHASE_TRANSFER :
        //rxn_gpu_SIMPOL_phase_transfer_calc_deriv_contrib(rate_constants,
        //        state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_TROE :
        rxn_gpu_troe_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_WET_DEPOSITION :
        rxn_gpu_wet_deposition_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;
      case RXN_ZSR_AEROSOL_WATER :
        rxn_gpu_ZSR_aerosol_water_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
        break;

        //case RXN_ARRHENIUS :
          //rxn_gpu_arrhenius_calc_deriv_contrib(rate_constants,
          //        state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);

//        rxn_gpu_tmp_arrhenius(
//                model_data, deriv_data, rxn_data, float_data, time_step, n_rxn);

        //break;
    }
  }
  __syncthreads();

  if (threadIdx.x < deriv_length)
  deriv[index] = deriv_data2[index];

  //if (threadIdx.x < deriv_length) {
  //  for (int i_spec = threadIdx.x; i_spec < deriv_length; i_spec += blockDim.x)//IT WORK for large rxn input data
  //    atomicAdd(&(deriv[i_spec]), deriv_data[i_spec]);
  //}
}

void rxn_calc_deriv_gpu(ModelDatagpu *model_data, N_Vector deriv, realtype time_step) {

  // Get a pointer to the derivative data
  int n_cells = model_data->n_cells;
  realtype *deriv_data = N_VGetArrayPointer(deriv);
  int *rxn_data = (int *) (model_data->rxn_data);
  int n_rxn_total_threads = rxn_data[0]*n_cells;
  double *state = model_data->state;
  double *rate_constants = model_data->rate_constants;

  /*if(countergpu==29) {
    for (int i_rxn=1; i_rxn<n_rxn; i_rxn++) {
      if(rxn_type==RXN_AQUEOUS_EQUILIBRIUM){
        printf(" type1: %d  ", int_data);
      }
    }
  }
  countergpu++;*/

  //memcpy(state_cpu, state, state_size);
  //HANDLE_ERROR(hipMemcpy(state_gpu, state, state_size, hipMemcpyHostToDevice));//Slower, use for large values

  //mdgpu = model_data;
  //Faster, use for few values
  state_gpu= state;
  //rate_constants_gpu= rate_constants;

  //Test to solve a bug with operations
  //rxn_gpu_tmp_arrhenius << < (n_rxn + MAX_N_GPU_THREAD - 1) / MAX_N_GPU_THREAD, MAX_N_GPU_THREAD >> >
   //(
   //mdgpu, derivgpu_data,int_pointer_gpu, double_pointer_gpu, time_step, n_rxn

   //     mdgpu, state, derivgpu_data, time_step, NV_LENGTH_S(deriv),
   //     n_rxn, int_pointer_gpu, double_pointer_gpu, int_max_size, double_max_size
    //);

  solveRxnBlock << < (n_rxn_total_threads + MAX_N_GPU_THREAD - 1) / MAX_N_GPU_THREAD, MAX_N_GPU_THREAD >> >
    (mdgpu, state_gpu, derivgpu_data, time_step, NV_LENGTH_S(deriv), model_data->n_state_var*n_cells,
    n_rxn_total_threads, n_cells, int_pointer_gpu, double_pointer_gpu, rate_constants_gpu);

  hipDeviceSynchronize();//retrieve errors (But don't retrieve anything for me)

  //HANDLE_ERROR(hipMemcpy(deriv_data, derivgpu_data, deriv_size, hipMemcpyDeviceToHost));//0.5secs

  //TODO: Avoid pinned memory for large cells maybe
  HANDLE_ERROR(hipMemcpy(deriv_cpu, derivgpu_data, deriv_size, hipMemcpyDeviceToHost));//0.29secs
  memcpy(deriv_data, deriv_cpu, deriv_size); //This is so fast(0.01secs)
}

void free_gpu_cu() {

  //HANDLE_ERROR( hipHostFree( derivgpu_data ) );
  //HANDLE_ERROR( hipHostFree( mdgpu ) );

  HANDLE_ERROR(hipFree( int_pointer_gpu ));
  HANDLE_ERROR(hipFree( double_pointer_gpu ));
  HANDLE_ERROR(hipFree(derivgpu_data));
  HANDLE_ERROR(hipFree(dev_start_rxn_param));
  HANDLE_ERROR(hipFree(rate_constants_gpu));

  //HANDLE_ERROR(hipFree(state_gpu)); //Invalid device pointer
  //HANDLE_ERROR(hipFree(env_gpu));
  //

  free(start_rxn_param);

  //free(int_pointer_gpu); //DO not, segmentation fault
  //free(double_pointer_gpu);//DO not, segmentation fault
  //free(deriv_cpu); DO not, segmentation fault
  //hipFree( derivgpu_data ); //TODO: ESTO PETA
  //hipFree( mdgpu );
  //HANDLE_ERROR( hipHostFree( deriv ) );

}

/*
void solveRxncpu(ModelDatagpu *model_data, double *deriv_data,
                   double time_step, int *int_data, double *float_data, int deriv_length, int n_rxn)
{

  int rxn_type = int_data[0];
  int *rxn_data = (int *) &(int_data[1]);

  switch (rxn_type) {
    case RXN_AQUEOUS_EQUILIBRIUM :
      rxn_cpu_aqueous_equilibrium_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_ARRHENIUS :
      rxn_cpu_arrhenius_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_CMAQ_H2O2 :
      rxn_cpu_CMAQ_H2O2_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_CMAQ_OH_HNO3 :
      rxn_cpu_CMAQ_OH_HNO3_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_CONDENSED_PHASE_ARRHENIUS :
      rxn_cpu_condensed_phase_arrhenius_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_EMISSION :
      rxn_cpu_emission_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_FIRST_ORDER_LOSS :
      rxn_cpu_first_order_loss_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_HL_PHASE_TRANSFER :
//rxn_cpu_HL_phase_transfer_calc_deriv_contrib(rate_constants,
//        model_data, state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_PDFITE_ACTIVITY :
      rxn_cpu_PDFiTE_activity_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_PHOTOLYSIS :
      rxn_cpu_photolysis_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_SIMPOL_PHASE_TRANSFER :
//rxn_cpu_SIMPOL_phase_transfer_calc_deriv_contrib(rate_constants,
//        model_data, state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_TROE :
      rxn_cpu_troe_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_WET_DEPOSITION :
      rxn_cpu_wet_deposition_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;
    case RXN_ZSR_AEROSOL_WATER :
      rxn_cpu_ZSR_aerosol_water_calc_deriv_contrib(rate_constants,
                state, deriv_data, (void *) rxn_data, float_data, time_step, deriv_length, n_rxn);
      break;

  }

}

void rxn_calc_deriv_cpu(ModelDatagpu *model_data, N_Vector deriv, realtype time_step)
{

  realtype *deriv_data = N_VGetArrayPointer(deriv);
  int *rxn_data = (int *) (model_data->rxn_data);
  int n_rxn = rxn_data[0];

  //Case i_rxn=0
  solveRxncpu(model_data, deriv_data, time_step, int_pointer, double_pointer, NV_LENGTH_S(deriv), n_rxn);

  for (int i_rxn = 1; i_rxn < n_rxn; i_rxn++) {

    int *int_data = (int *) &(((int *) int_pointer)[int_sizes[i_rxn - 1]]);
    double *float_data = (double *) &(((double *) double_pointer)[double_sizes[i_rxn - 1]]);

    solveRxncpu(model_data, deriv_data, time_step, int_data, float_data, NV_LENGTH_S(deriv), n_rxn);
  }
}
*/


}
